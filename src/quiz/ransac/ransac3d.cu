
#include <hip/hip_runtime.h>
__global__ void add(int *a, int *b, int *c) {
    int index = threadIdx.x;
    c[index] = a[index] + b[index];
}